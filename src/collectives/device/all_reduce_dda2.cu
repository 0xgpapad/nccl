#include "hip/hip_runtime.h"
// (c) Meta Platforms, Inc. and affiliates. Confidential and proprietary.

#include "all_reduce.h"
#include "all_reduce_dda2.h"
#include "collectives.h"
#include "common.h"

#define idx(nranks, i, j) ((i) * (nranks) + (j))

template <typename T>
static inline __device__ uint32_t
vecElementAdd(const uint32_t& a, const uint32_t& b) {
  if (std::is_same<T, half>::value) {
    const __half* x = reinterpret_cast<const __half*>(&a);
    const __half* y = reinterpret_cast<const __half*>(&b);

#if (__CUDA_ARCH__ >= 700)
    __half2 p = __halves2half2(x[0], x[1]);
    __half2 q = __halves2half2(y[0], y[1]);

    __half2 z = __hadd2(p, q);
    return (reinterpret_cast<uint32_t*>(&z))[0];
#else
    half z[2] = { __hadd(x[0], y[0]), __hadd(x[1], y[1]) };
    return (reinterpret_cast<uint32_t*>(z))[0];
#endif

#if defined(__CUDA_BF16_TYPES_EXIST__)
  } else if (std::is_same<T, __hip_bfloat16>::value) {
    const __hip_bfloat16* x = reinterpret_cast<const __hip_bfloat16*>(&a);
    const __hip_bfloat16* y = reinterpret_cast<const __hip_bfloat16*>(&b);

#if (__CUDA_ARCH__ >= 800)
    __hip_bfloat162 p = {x[0], x[1]};
    __hip_bfloat162 q = {y[0], y[1]};

    __hip_bfloat162 z = __hadd2(p, q);
    return (reinterpret_cast<uint32_t*>(&z))[0];
#else
    __hip_bfloat16 z[2] = {x[0] + y[0], x[1] + y[1]};
    return (reinterpret_cast<uint32_t*>(z))[0];
#endif
  }
#endif

  return 0;
}

/* create a special version of seqAdd that can be disabled at
 * compile-time for bfloat16 (using enable_if).  This is because the
 * base version of seqAdd does not compile for bfloat16, so we are
 * essentially tricking the compiler.  We never call this version for
 * bfloat16, so it doesn't matter that it does not compile, but the
 * compiler unfortunately does not know that. */
template <typename T, uint32_t NRANKS>
static inline __device__
typename std::enable_if<!std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
    && !std::is_same<T, __hip_bfloat16>::value
#endif
    , uint4>::type
seqAdd(const T** src, size_t offset) {
  T dst[16 / sizeof(T)] = {0};
  for (int i = 0; i < NRANKS; i++) {
    uint4 vals = reinterpret_cast<const uint4*>(&src[i][offset])[0];
    const T* src_d = reinterpret_cast<const T*>(&vals);
    for (int j = 0; j < 16 / sizeof(T); j++) {
      dst[j] += src_d[j];
    }
  }
  return reinterpret_cast<uint4*>(&dst)[0];
}

template <typename T, uint32_t NRANKS>
static inline __device__
typename std::enable_if<std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
    || std::is_same<T, __hip_bfloat16>::value
#endif
    , uint4>::type
seqAdd(const T** src, size_t offset) {
  uint4 x = {0, 0, 0, 0};

  return x;
}

template <typename T, uint32_t NRANKS>
static inline __device__ uint4 vecAdd(const T** src, size_t offset) {
  if (std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
      || std::is_same<T, __hip_bfloat16>::value
#endif
  ) {
    uint4 dst = {0, 0, 0, 0};
    for (int i = 0; i < NRANKS; i++) {
      /* 16-byte load */
      uint4 vals = reinterpret_cast<const uint4*>(&src[i][offset])[0];

      /* vector additions */
      dst.x = vecElementAdd<T>(dst.x, vals.x);
      dst.y = vecElementAdd<T>(dst.y, vals.y);
      dst.z = vecElementAdd<T>(dst.z, vals.z);
      dst.w = vecElementAdd<T>(dst.w, vals.w);
    }
    return dst;
  } else {
    return seqAdd<T, NRANKS>(src, offset);
  }
}

template <typename T>
static inline __device__
typename std::enable_if<std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
    || std::is_same<T, __hip_bfloat16>::value
#endif
    , uint4>::type
vecAdd(const T* src_a, const T* src_b) {
  /* 16-byte loads */
  uint4 vals_a = reinterpret_cast<const uint4*>(src_a)[0];
  uint4 vals_b = reinterpret_cast<const uint4*>(src_b)[0];

  /* vector additions */
  uint4 dst;
  dst.x = vecElementAdd<T>(vals_a.x, vals_b.x);
  dst.y = vecElementAdd<T>(vals_a.y, vals_b.y);
  dst.z = vecElementAdd<T>(vals_a.z, vals_b.z);
  dst.w = vecElementAdd<T>(vals_a.w, vals_b.w);
  return dst;
}

template <typename T>
static inline __device__
typename std::enable_if<!std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
    && !std::is_same<T, __hip_bfloat16>::value
#endif
    , uint4>::type
vecAdd(const T* src_a, const T* src_b) {
  /* 16-byte loads */
  uint4 vals_a = reinterpret_cast<const uint4*>(src_a)[0];
  uint4 vals_b = reinterpret_cast<const uint4*>(src_b)[0];

  /* cast back to original type and do sequential additions */
  T dst[16 / sizeof(T)];
  const T* src_a_loaded = reinterpret_cast<const T*>(&vals_a);
  const T* src_b_loaded = reinterpret_cast<const T*>(&vals_b);
  for (int j = 0; j < 16 / sizeof(T); j++) {
    dst[j] = src_a_loaded[j] + src_b_loaded[j];
  }
  return reinterpret_cast<uint4*>(&dst)[0];
}

template <uint32_t NRANKS>
static inline __device__ void
barrier(uintptr_t* barrierMbox, uintptr_t barrierFlag, int rank) {
  volatile uintptr_t* barrier_d = barrierMbox;
  const int gtidx = threadIdx.x + blockDim.x * blockIdx.x;

  if (gtidx == 0) {
    barrier_d[rank] = barrierFlag;
  }

  if (threadIdx.x < NRANKS) {
    while ((barrier_d[threadIdx.x] & 1UL) != (barrierFlag & 1UL)) {
    }
  }

  /* remaining threads in the block wait */
  __syncthreads();
}

/* We use a simple Allgather + local reduce algorithm here.  For small
 * messages, we are mostly latency bound on fast networks such as
 * NVLink.  So fetching data from all the GPUs simultaneously should
 * basically take the same amount of time as fetching data from one
 * GPU.  This algorithm directly reads data from the other GPUs and
 * reduces it into the local destination buffer. */

template <typename T, uint32_t NRANKS>
__global__ void ncclKernel_AllReduce_DDA2_Flat(
    uintptr_t barrierFlag,
    DdaDeviceState* devStates,
    int rank,
    const T* sendbuff,
    T* recvbuff,
    size_t count) {
  const int gtidx = blockDim.x * blockIdx.x + threadIdx.x;

  // always use rank0's barrierMbox as the shared barrier
  uintptr_t* mbox = devStates[0].barrierMbox;
  barrier<NRANKS>(mbox, (reinterpret_cast<uintptr_t>(sendbuff)) | barrierFlag, rank);

  const T* src[NRANKS];
  for (int i = 0; i < NRANKS; i++) {
    int r = (rank + i) & (NRANKS - 1);
    src[i] = reinterpret_cast<const T*>(mbox[r] & ~1UL);
  }

  for (size_t offset = gtidx * 16 / sizeof(T); offset < count;
       offset += gridDim.x * blockDim.x * 16 / sizeof(T)) {
    reinterpret_cast<uint4*>(&recvbuff[offset])[0] =
      vecAdd<T, NRANKS>(src, offset);
  }

  barrier<NRANKS>(mbox + NRANKS, barrierFlag, rank);
}

DECL_DDA2_FUNC(char);
DECL_DDA2_FUNC(uint8_t);
DECL_DDA2_FUNC(int32_t);
DECL_DDA2_FUNC(uint32_t);
DECL_DDA2_FUNC(int64_t);
DECL_DDA2_FUNC(uint64_t);
DECL_DDA2_FUNC(half);
DECL_DDA2_FUNC(float);
DECL_DDA2_FUNC(double);
#if defined(__CUDA_BF16_TYPES_EXIST__)
DECL_DDA2_FUNC(__hip_bfloat16);
#endif
